// This program implements tiled 1D conv using shared memory


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <cassert>
#include <cstdlib>

#define MASK_LENGTH 7
__constant__ int mask[MASK_LENGTH];


__global__ void convolution_1d(int *array, int *result, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Store all elements needed to compute output in shared memory
    extern __shared__ int s_array[];

    int r = MASK_LENGTH / 2;
    int d = 2 * r; // total number of padded elements
    int n_padded = blockDim.x + d; // size of the padded shared memory array
    int offset = threadIdx.x + blockDim.x; // offset for second set of loads in shared memory
    int g_offset = blockDim.x * blockIdx.x + offset; // Global offset for the array in DRAM

    // Load the lower elements first starting at the halo
    // This ensures divergence only once
    s_array[threadIdx.x] = array[tid];

    // Load the remaining upper elements
    if (offset < n_padded){
        s_array[offset] = array[g_offset];
    }
    __syncthreads();

    int temp = 0;
    for (int j = 0; j < MASK_LENGTH; j++){
        temp += s_array[threadIdx.x + j] * mask[j];
    }
    result[tid] = temp;
}

// updated: We have now zero padded array!
void verify_result(int *array, int *mask, int *result, int n) {
  int temp;
  for (int i = 0; i < n; i++) {
    temp = 0;
    for (int j = 0; j < MASK_LENGTH; j++) {
      temp += array[i + j] * mask[j];
    }
    assert(temp == result[i]);
  }
}

int main(){
    // Number of elements in result array
    int n = 1 << 20;

    // Size of the array in bytes
    int bytes_n = n * sizeof(int);

    // Size of the mask in bytes
    size_t bytes_m = MASK_LENGTH * sizeof(int);

    // Radius for padding the array
    int r = MASK_LENGTH / 2;
    int n_p = n + r * 2;

    // Size of the padded array in bytes
    size_t bytes_p = n_p * sizeof(int);

    // Allocate the array (include edge elements)...
    int *h_array = new int[n_p];

    // ... and initialize it
    for (int i = 0; i < n_p; i++) {
        if ((i < r) || (i >= (n + r))) {
        h_array[i] = 0;
        } else {
        h_array[i] = rand() % 100;
        }
    }

    // Allocate the mask and initialize it
    int *h_mask = new int[MASK_LENGTH];
    for (int i = 0; i < MASK_LENGTH; i++) {
        h_mask[i] = rand() % 10;
    }

    // Allocate space for the result
    int *h_result = new int[n];

    // Allocate space on the device
    int *d_array, *d_result;
    hipMalloc(&d_array, bytes_p);
    hipMalloc(&d_result, bytes_n);

    // Copy the data to the device
    hipMemcpy(d_array, h_array, bytes_p, hipMemcpyHostToDevice);

    // Copy the mask directly to the symbol
    // This would require 2 API calls with cudaMemcpy
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    // Threads per TB
    int THREADS = 256;

    // Number of TBs
    int GRID = (n + THREADS - 1) / THREADS;

    // Amount of space per-block for shared memory
    // This is padded by the overhanging radius on either side
    size_t SHMEM = (THREADS + r * 2) * sizeof(int);

    // Call the kernel
    convolution_1d<<<GRID, THREADS, SHMEM>>>(d_array, d_result, n);

    // Copy back the result
    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    // Verify the result
    verify_result(h_array, h_mask, h_result, n);

    std::cout << "COMPLETED SUCCESSFULLY\n";

    // Free allocated memory on the device and host
    delete[] h_array;
    delete[] h_result;
    delete[] h_mask;
    hipFree(d_result);

    return 0;
}