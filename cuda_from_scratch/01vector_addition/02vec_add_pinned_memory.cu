// Vector addition (pinned_memory)
#include <hip/hip_runtime.h>

#include <math.h>
#include <algorithm>
#include <iostream>
#include <vector>
#include <assert.h>

using std::begin;
using std::copy;
using std::cout;
using std::end;
using std::generate;
using std::vector;

// Cuda kernel for vector addition
__global__ void vecAdd(int* a, int* b, int* c, int N) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}

}

// check result
void verify_result(int* a, int* b, int* c, int N) {
	for (int i = 0; i < N; i++) {
		assert(c[i] == a[i] + b[i]);
	}

}





int main() {

	// Initialize a vector with random numbers
	constexpr int N = 1 << 16;
	constexpr size_t bytes = sizeof(int) * N;
	
	// Allocate memory on CPU
	int* h_a, * h_b, * h_c;
	hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
	hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
	hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

	// initialize
	for (int i = 0; i < N; i++) {
		h_a[i] = rand() % 100;
		h_b[i] = rand() % 100;
	}

	// Allocate memory on GPU
	int* d_a, * d_b, * d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Copy data from host to Device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// THREAD
	int NUM_THREADS = 1 << 10;
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	vecAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Verify calculation
	verify_result(h_a, h_b, h_c, N);

	// Free Memory
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout << "Completed Successfully.\n";

	return 0;

}