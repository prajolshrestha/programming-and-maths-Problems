#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <algorithm>
#include <assert.h>
#include <vector>
#include <iterator>
#include <cstdlib>

__global__ void vecAdd(int* a, int* b, int* c, int N) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}

}

void init_vector(int* a, int* b, int N) {
	for (int i = 0; i < N; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;

	}
}

void verify_results(int* a, int* b, int* c, int N) {
	for (int i = 0; i < N; i++) {
		assert(c[i] == a[i] + b[i]);
	}
}


int main() {

	constexpr int N = 1 << 16;
	size_t bytes = sizeof(int) * N;

	// Method 0: Baseline
	
	// CPU
	std::vector<int> g(N);
	std::vector<int> h(N);
	std::vector<int> i(N);
	
	//initialize
	std::generate(std::begin(g), std::end(g), []() {return std::rand() % 100; });
	std::generate(std::begin(h), std::end(h), []() {return std::rand() % 100; });

	// GPU
	int* d_g, * d_h, * d_i;
	hipMalloc(&d_g, bytes);
	hipMalloc(&d_h, bytes);
	hipMalloc(&d_i, bytes);

	hipMemcpy(d_g, g.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_h, h.data(), bytes, hipMemcpyHostToDevice);

	// Threads
	int NUM_THREADS = 1 << 10;
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	vecAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_g, d_h, d_i, N);

	hipMemcpy(i.data(), d_i, bytes, hipMemcpyDeviceToDevice);

	

	hipFree(d_g);
	hipFree(d_h);
	hipFree(d_i);




	// Method 1: Pinned Memory ///////////////////////////////////////////////
	
	// CPU
	int* a, * b, * c;
	hipHostMalloc(&a, bytes);
	hipHostMalloc(&b, bytes);
	hipHostMalloc(&c, bytes);

	init_vector(a, b, N); // initialize

	// GPU
	int* d_a, * d_b, * d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

	// Work on threads
	//int NUM_THREADS = 1 << 10;
	//int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	vecAdd<<<NUM_BLOCKS, NUM_THREADS>>>(a, b, c, N); // Kernel call

	// CPU
	hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

	verify_results(a, b, c, N);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	// Method 2:Unified Memory (prefetch) /////////////////////////////////////////////////////

	int id = hipGetDevice(&id); // get device id for cuda calls

	// CPU
	int* d, * e, *f;
	hipMallocManaged(&d, bytes);
	hipMallocManaged(&e, bytes);
	hipMallocManaged(&f, bytes);
	
	init_vector(d, e, N);

	// GPU [Transfer not required, done automatically]
	// Thread
	int BLOCK_SIZE = 1 << 10;
	int GRID_SIZE = (int)ceil(N / BLOCK_SIZE);

	vecAdd<<<GRID_SIZE, BLOCK_SIZE>>>(d, e, f, N);

	// prefetch for performance boost
	hipMemPrefetchAsync(d, bytes, id);
	hipMemPrefetchAsync(e, bytes, id);

	hipDeviceSynchronize(); // synchronize 

	hipMemPrefetchAsync(f, bytes, hipCpuDeviceId); // prefetch

	verify_results(d, e, f, N);


	
	return 0;



}