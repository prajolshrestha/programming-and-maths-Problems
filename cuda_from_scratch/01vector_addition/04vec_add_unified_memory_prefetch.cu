#include "hip/hip_runtime.h"
// Vector addition using unified memory with prefetch[virtual memory]
// Instead of having seperate device memory and host memory having to manage both these independently,
// Some other system takes care of it. All I worry about is my kernel code. 
// ie, we dont have to manage memory

// To increase performance we use prefetch

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>


__global__ void vecAddUM(int* a, int* b, int* c, int N) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

void init_vector(int* a, int* b, int N) {
	for (int i = 0; i < N; i++) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}
}

void check_answer(int* a, int* b, int* c, int N) {
	for (int i = 0; i < N; i++) {
		assert(c[i] == a[i] + b[i]);
	}
}


int main() {
	// get the device Id for other CUDA calls
	int id = hipGetDevice(&id);

	int N = 1 << 16;
	size_t bytes = sizeof(int) * N;

	// Declare unified memory pointers and initialize them with random numbers
	int* a, * b, * c;
	// allocate memory
	hipMallocManaged(&a, bytes); // Transfer of data betn CPU and GPU happen automatically
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);
	init_vector(a, b, N);

	// Thread
	int BLOCK_SIZE = 256;
	int GRID_SIZE = (int)ceil(N / BLOCK_SIZE);

	// call cuda Kernel
	// Performance boost:
	// Problem: GPU starts up and says I dont have any of the data. Page Fault ( page in memory from CPU to GPU) [page size: CPU is of 4kilobytes]
	// When data should be where? ==> Prefetching : Behind the scene you can transfer data while you do other things. (ahead of time)
	hipMemPrefetchAsync(a, bytes, id);
	hipMemPrefetchAsync(b, bytes, id);
	vecAddUM << <GRID_SIZE, BLOCK_SIZE >> > (a, b, c, N);

	// wait for all previous operations before using values
	hipDeviceSynchronize();

	hipMemPrefetchAsync(c, bytes, hipCpuDeviceId); // Prefetching device memory back to the CPU

	// check result
	check_answer(a, b, c, N);

	printf("COmpleted Suffessfully\n");

	return 0;

}