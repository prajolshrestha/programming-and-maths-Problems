#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>


// Initiate a vector
void vector_init(float *a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = (float)(rand() % 100);
    }
}

// verify the result
void verify_result (float *a, float *b, float *c, float factor, int n) {
    for (int i = 0; i < n; i++) {
        assert(c[i] == factor * a [i] + b[i]);
    }
}


// nvcc -o cublas_vecadd 05cuBLAS_vec_add.cu -lcublas -lcurand
int main() {
    int n = 1 << 2;
    size_t bytes = n * sizeof(float);

    // Declare vector pointers
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    // Allocate memory
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);


    // Initialize vectors
    vector_init(h_a, n);
    vector_init(h_b, n);

    // create and initialize a new context
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // copy the vectors over to the device
    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    // Lunch simple saxpy kernel (single precision a * x + y)
    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

    // copy the result vector back out
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    // Print out the result
    verify_result(h_a, h_b, h_c, scale, n);

    // clearn up the created handle
    hipblasDestroy(handle);

    // Release allocated memory
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    printf("Completed Successfully\n");

    return 0;

}