
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <assert.h>
#include <math.h>

// Static shmem calculation for convenience (int 16 x 16 matrix) |Note: We can also allocate dynamically!
#define SHMEM_SIZE 16 * 16 * 4 // 256 Threads per thread block

// Kernel
__global__ void tiledMatrixMul(int *a, int *b, int *c, int n, int tile_size) {
    // Two statically-sized pieces of shared memory
    __shared__ int A[SHMEM_SIZE]; // This declared array will be stored in a shared memory!
    __shared__ int B[SHMEM_SIZE];

    // Shorten these parameters for clean re-use
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Calculate global row and column positions for this thread
    int row = by * tile_size + ty; // tile_size = blockDim = block size
    int col = bx * tile_size + tx;

    // Intermediate sum for elements being written
    int temp_val = 0;

    // Sweep tiles over entire matrix
    for (int i=0; i < (n /tile_size); i++){ // Important step: We go block by block
        // Transfer one-one block of data to shared memory for efficient access
        A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)]; // One (block) row of data stored in A
        B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col]; // one (block) col of data

        // Ensure all threads have loaded their data before proceeding
        __syncthreads();

        // Actual calculation (Two blocks multiplied at a time))
        for (int j = 0; j < tile_size; j++) {
            temp_val += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
        }

        // Ensure some threads don't progress and stomp current shared memory values
        __syncthreads();
    }
    c[(row * n) + col] = temp_val;
}

void check_answer(int *a, int *b, int *c, int n) {
    int *verify_c;
    verify_c = (int*)malloc(n * n * sizeof(int));
    int temp_val;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            temp_val = 0;
            for (int k = 0; k < n; k++) {
                temp_val += a[i * n + k] * b[k * n + j];
            }
            verify_c[i * n + j] = temp_val;
        }
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            assert(c[i * n + j] == verify_c[i * n + j]);
        }
    }
}

void init_matrix(int *a, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i * n + j] = rand() % 10;
        }
    }
}

int main() {
    int n = 1 << 10; // 1024 x 1024
    size_t bytes = n * n * sizeof(int);

    // host matrix pointers
    int *h_a, *h_b, *h_c;

    // device matrix pointers
    int *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialize matrices
    init_matrix(h_a, n);
    init_matrix(h_b, n);

    // copy matrixes to the device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threads per block
    int BLOCK_SIZE = 16;

    // Blocks in each dimension
    int GRID_SIZE = (int)ceil(n / BLOCK_SIZE);

    // use dim3 objects for 2-D grids and threadblocks
    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    // Lunch kernel
    tiledMatrixMul <<<grid, threads>>> (d_a, d_b, d_c, n, BLOCK_SIZE);

    // copy result back from device
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // verify the result
    check_answer(h_a, h_b, h_c, n);

    // free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    printf("Completed Successfully");

    return 0;



}